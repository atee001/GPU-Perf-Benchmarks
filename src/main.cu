#include <iostream>
#include <cassert>
#include <hip/hip_runtime.h>

#include "../include/Benchmark.cuh"

void printDeviceProperties(int deviceIndex) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, deviceIndex);
    
    checkError();
    std::cout << "Device " << deviceIndex << ": " << prop.name << std::endl;
    std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "  Total global memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "  Shared memory per block: " << prop.sharedMemPerBlock / 1024 << " KB" << std::endl;
    std::cout << "  Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "  Max thread dimensions: (" 
              << prop.maxThreadsDim[0] << ", "
              << prop.maxThreadsDim[1] << ", "
              << prop.maxThreadsDim[2] << ")" << std::endl;
    std::cout << "  Max grid size: (" 
              << prop.maxGridSize[0] << ", "
              << prop.maxGridSize[1] << ", "
              << prop.maxGridSize[2] << ")" << std::endl;
    std::cout << "  Warp size: " << prop.warpSize << std::endl;
    std::cout << "  Multi-processor count: " << prop.multiProcessorCount << std::endl;
    std::cout << "  L2 cache size: " << prop.l2CacheSize / 1024 << " KB" << std::endl;
    std::cout << "  Max register count per block: " << prop.regsPerBlock << std::endl;

    std::cout << "  Number of streaming multiprocessors (SMs): " << prop.multiProcessorCount << std::endl;
}

int main(int argc, char* argv[])
{

	if(argc < 3)
	{
		std::cout << "Expected Input <.exe> <i> <num_iter>" << std::endl;
		return 1;
	}
	
	printDeviceProperties(0);
	unsigned int i = (unsigned int)atoi(argv[1]);
	std::cout << "i " << i << std::endl;	
	size_t num_b = (1UL << 32); //defualt do 4 GB 
	param_t pType;
	pType.arrSize = (1UL << 10) << i; 
	pType.numAccess = num_b / pType.arrSize; 
	pType.blockSize = 512;
        pType.gridSize = 1024 * 4;
	std::cout << "Block Size " << pType.blockSize 
	<< "Grid Size " << pType.gridSize 
	<< "Array Size " << pType.arrSize
	<< "Num Access " << pType.numAccess << std::endl;	


	readBWTest<unsigned int>(&pType);	
	//std::cout << "Hello World!" << std::endl;
	return 0;
}
